#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <iomanip>
#define N 1000
using namespace std;

__global__ void mattmult(double* a, double* b, double* c)
{
	//глобальные координаты
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	//локальные
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	double sum = 0;
	int end = ((int)N / 32) + 1;
	for (int i = 0;i < end;i++)
	{
		__syncthreads();
		__shared__ double casheA[32][32];
		__shared__ double casheB[32][32];
		casheA[tx][ty] = a[x + N * ty + 32 * i * N];
		casheB[tx][ty] = b[tx + y * N + 32 * i];
		__syncthreads();
		for (int k = 0; k < 32; k++)
			sum += casheA[tx][k] * casheB[k][ty];
		
	}
	if (x < N && y < N)
		c[x + y * N] = sum;
}


int main()
{
	unsigned int mem_size = sizeof(double) * N * N;
	double* hA, * hB, * hC;
	double* da, * db, * dc;
	int threadsPerBlock = 32;
	int blocksPerGrid = int((N / ((float)threadsPerBlock)) + 1);
	dim3 blockDim(threadsPerBlock, threadsPerBlock, 1);
	dim3 gridDim(blocksPerGrid, blocksPerGrid, 1);
	hipHostAlloc((void**)&hA, mem_size, hipHostMallocDefault);
	hipHostAlloc((void**)&hB, mem_size, hipHostMallocDefault);
	hipHostAlloc((void**)&hC, mem_size, hipHostMallocDefault);

	hipMalloc((void**)&da, mem_size);
	hipMalloc((void**)&db, mem_size);
	hipMalloc((void**)&dc, mem_size);

	for (int i = 0;i <= (N * N - 1);i++)
	{
		hA[i] = rand() % 100;
		hB[i] = rand() % 100;
		hC[i] = 0;
	}
	hipStream_t stream[2];
	for (int i = 0; i < 2; ++i) hipStreamCreate(&stream[i]);

	hipEvent_t start, stop;
	float gpu_time = 0.0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	//hipMemcpy(da, hA, mem_size , hipMemcpyHostToDevice);
	//hipMemcpy(db, hB, mem_size , hipMemcpyHostToDevice);

	hipMemcpyAsync(da, hA, mem_size, hipMemcpyHostToDevice, stream[0]);
	hipMemcpyAsync(db, hB, mem_size, hipMemcpyHostToDevice, stream[1]);

	hipDeviceSynchronize();
	mattmult << < gridDim, blockDim>> > (da, db, dc);
	for (int i = 0; i < 2; i++)
	{
		int offset = int(N * N / 2);
		hipMemcpyAsync(hC + offset * i, dc + offset * i, mem_size / 2, hipMemcpyDeviceToHost, stream[i]);
	}
	//hipMemcpy(hC, dc, mem_size, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpu_time, start, stop);
	cout << "Time GPU=" << gpu_time << " ms";
	for (int i = 0; i < 2; ++i) hipStreamDestroy(stream[i]);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	cout << "difference =" ;
	int dif = 0;
	for (int i =0 ;i <N;i++)
	{
		for (int j =0 ;j <N ;j++)
		{
			double s = 0;
			for (int k = 0;k < N;k++)
			{
				s += hA[i + k * N] * hB[k + j * N];
			}	
			dif += abs(s - hC[i + j * N]);
		}
	}
	cout  << dif << endl;
	hipFree(da);
	hipFree(db);
	hipFree(dc);
	hipHostFree(hA);
	hipHostFree(hB);
	hipHostFree(hC);
	/*printf("\nYour matrix A=");
	for (int i = 0;i < N;i++) {
		printf("\n");
		for (int j = 0;j < N;j++) {
			printf("%0.2lf ", hA[i + j*N]);
		}
	}
	printf("\nYour matrix B=");
	for (int i = 0;i < N;i++) {
		printf("\n");
		for (int j = 0;j < N;j++) {
			printf("%0.2lf ", hB[i + j * N]);
		}
	}
	printf("\nYour matrix C=");
		for (int i = 0;i < N;i++) {
			printf("\n");
			for (int j = 0;j < N;j++) {
				printf("%0.2lf ", hC[i + j * N]);
			}
		}
	*/

}




